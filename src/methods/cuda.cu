#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>


#include "../matrix/matrix.h"
#include "../matrix/csr.h"
#include "../matrix/ellpack.h"
#include "../io/iomanager.h"
#include "../utils/utils.h"


unsigned int vm_thr_block = 512, scalar_thr_block = 512, n_blocks_vm, n_blocks_scalar;


void cudaCheckError(int line) {
	hipError_t e = hipGetLastError();
	if(e != hipSuccess) {
		printf("Cuda failure %s:%d: '%s'\n", __FILE__, line, hipGetErrorString(e));
	}
}



__global__ void scalarCSR(int * m, int * irp, int * ja, double * as, double * x, double * y) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < *m) {
		double temp = 0.0;
		for (int j = irp[i]; j < irp[i + 1]; ++j) {
			temp += as[j] * x[ja[j]];
		}
		y[i] = temp;
	}
}

__global__ void vectorMiningCSR(int * m, int * d_warp_size, int * irp, int * ja, double * as, double * x, double * y) {
	extern __shared__ volatile double sdata[];
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int tid = threadIdx.x;
	int warp_size = *d_warp_size;

	int warp = i / warp_size; 
	int lane = i & (warp_size - 1); 
	int row = warp;

	sdata[tid] = 0;

	if (row < *m) {
		for (int j = irp[row] + lane ; j < irp[row + 1]; j += warp_size)
			sdata[tid] += as[j] * x[ja[j]];

		if (warp_size == 32) { if (lane < 16) { sdata[tid] += sdata[tid + 16]; } }
		if (lane < 8) { sdata[tid] += sdata[tid + 8]; }
		if (lane < 4) { sdata[tid] += sdata[tid + 4]; }
		if (lane < 2) { sdata[tid] += sdata[tid + 2]; }
		if (lane < 1) { sdata[tid] += sdata[tid + 1]; }

		if (lane == 0)
			y[row] += sdata[tid];
	}
}

__global__ void scalarEllpack(int * m, int * ja, double * as, double * x, double * y, int * maxnz) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int rows = *m, maximum_nz = *maxnz;
	if (i < rows) {
		double temp = 0.0;
		for (unsigned int j = 0; j < *maxnz; ++j) {
			temp += as[i * maximum_nz + j] * x[ja[i * maximum_nz + j]];
		}
		y[i] = temp;
	}
}


void allocateCSR(CSR * &csr, int * &irp, int * &ja, double * &as, double * &x, double *&y, int &m, int &n) {
	int nz = csr->getnz();

	hipMalloc((void**)&irp, sizeof(int) * (n + 1));
	hipMalloc((void**)&ja, sizeof(int) * nz);
	hipMalloc((void**)&as, sizeof(double) * nz);
	hipMalloc((void**)&x, sizeof(double) * n);
	hipMalloc((void**)&y, sizeof(double) * n);

	hipMemcpy(irp, csr->irp, sizeof(int) * (n + 1), hipMemcpyHostToDevice);
	hipMemcpy(ja, csr->getja(), sizeof(int) * nz, hipMemcpyHostToDevice);
	hipMemcpy(as, csr->getas(), sizeof(double) * nz, hipMemcpyHostToDevice);
	hipMemcpy(x, csr->getX(), sizeof(double) * n, hipMemcpyHostToDevice);
	hipMemcpy(y, csr->y, sizeof(double) * n, hipMemcpyHostToDevice);
}

void allocateEllpack(Ellpack * &ellpack, int * &ja, double * &as, double * &x, double * &y, int * &maxnz, int &m, int &n) {
	long long int host_maxnz = ellpack->getmaxnz();
	long long int rows = m;

	int * host_ja = ellpack->get1Dja();
	double * host_as = ellpack->get1Das();

	hipMalloc((void**)&ja, sizeof(int) * rows * host_maxnz);
	hipMalloc((void**)&as,  sizeof(double) * rows * host_maxnz);
	hipMalloc((void**)&x, sizeof(double) * rows);
	hipMalloc((void**)&y, sizeof(double) * rows);
	hipMalloc((void**)&maxnz, sizeof(int));
	hipMemcpy(ja, host_ja, sizeof(int) * rows * host_maxnz, hipMemcpyHostToDevice);
	hipMemcpy(as, host_as, sizeof(double) * rows * host_maxnz, hipMemcpyHostToDevice);
	hipMemcpy(x, ellpack->getX(), sizeof(double) * rows, hipMemcpyHostToDevice);
	hipMemcpy(y, ellpack->y, sizeof(double) * m, hipMemcpyHostToDevice);
	hipMemcpy(maxnz, &host_maxnz, sizeof(int), hipMemcpyHostToDevice);
}

void deallocateCSR(int * &irp, int * &ja, double * &as, double * &x, double *&y) {
	hipFree(irp);
	hipFree(ja);
	hipFree(as);
	hipFree(x);
	hipFree(y);
}

void deallocateEllpack(int * &ja, double * &as, double * &x, double * &y, int * &maxnz) {
	hipFree(ja);
	hipFree(as);
	hipFree(x);
	hipFree(y);
	hipFree(maxnz);
}

void collectResults(CSR * &csr, Ellpack * &ellpack, double * &csr_y, double * &ellpack_y, int &n) {
	hipMemcpy(csr->y, csr_y, sizeof(double) * n, hipMemcpyDeviceToHost);
	hipMemcpy(ellpack->y, ellpack_y, sizeof(double) * n, hipMemcpyDeviceToHost);
}

void getBlockNumbers(int m, int &warp_size) {
	n_blocks_scalar = m / scalar_thr_block;
	if (m % scalar_thr_block > 0.0) {
		n_blocks_scalar++;
	}
	n_blocks_vm = (m * warp_size) / vm_thr_block;
	if ((m * warp_size) % vm_thr_block > 0.0) {
		n_blocks_vm++;
	}
	if (n_blocks_vm > MAX_N_BLOCKS) {
		vm_thr_block *= 2;
		n_blocks_vm = (m * warp_size) / vm_thr_block;
		if ((m * warp_size) % vm_thr_block > 0.0) {
			n_blocks_vm++;
		}
	}
	if (n_blocks_vm > MAX_N_BLOCKS) {
		warp_size /= 2;
		n_blocks_vm = (m * warp_size) / vm_thr_block;
		if ((m * warp_size) % vm_thr_block > 0.0) {
			n_blocks_vm++;
		}
	}
}

void solveCuda(IOmanager * io, std::string path, CSR * &csr, Ellpack * &ellpack) {

	int m = csr->getRows();
	int n = csr->getCols();
	int warp_size = 32, *d_warp_size;
	
	StopWatchInterface* timer = 0;
	sdkCreateTimer(&timer);

	getBlockNumbers(m, warp_size);
	const int shmem_size = vm_thr_block * sizeof(double);
	
	int * csr_irp, * csr_ja, * ellpack_ja, * maxnz, * rows;
	double * csr_as, * csr_x, * csr_y, * ellpack_as, * ellpack_x, * ellpack_y;
	if (csr->fitsInMemory()) {
		allocateCSR(csr, csr_irp, csr_ja, csr_as, csr_x, csr_y, m, n);
	}
	if (ellpack->fitsInMemory()) {
		allocateEllpack(ellpack, ellpack_ja, ellpack_as, ellpack_x, ellpack_y, maxnz, m, n);
	}
	cudaCheckError(__LINE__);

	hipMalloc((void**)&rows, sizeof(int));
	hipMemcpy(rows, &m, sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_warp_size, sizeof(int));
	hipMemcpy(d_warp_size, &warp_size, sizeof(int), hipMemcpyHostToDevice);
	cudaCheckError(__LINE__);

	for (int k = 0; k < NR_RUNS + 2; ++k) {
		if (csr->fitsInMemory()) {

			timer->reset();
			timer->start();
			scalarCSR<<<n_blocks_scalar, scalar_thr_block>>>(rows, csr_irp, csr_ja, csr_as, csr_x, csr_y);
			hipDeviceSynchronize();
			timer->stop();
			csr->trackCSRTime(SCALAR, timer->getTime());
			timer->reset();

			hipMemset(csr_y, 0.0, sizeof(double) * m);
			cudaCheckError(__LINE__);

			timer->reset();
			timer->start();
			vectorMiningCSR<<<n_blocks_vm, vm_thr_block, shmem_size>>>(rows, d_warp_size, csr_irp, csr_ja, csr_as, csr_x, csr_y);
			hipDeviceSynchronize();
			timer->stop();
			csr->trackCSRTime(VECTOR_MINING, timer->getTime());
			timer->reset();
			cudaCheckError(__LINE__);
		}
		
		if (ellpack->fitsInMemory()) {
			timer->reset();
			timer->start();
			scalarEllpack<<<n_blocks_scalar, scalar_thr_block>>>(rows, ellpack_ja, ellpack_as, ellpack_x, ellpack_y, maxnz);
			hipDeviceSynchronize();
			timer->stop();
			ellpack->trackTime(timer->getTime());
			timer->reset();

			cudaCheckError(__LINE__);
		}

		if (k != NR_RUNS) {
			if (csr->fitsInMemory()) hipMemset(csr_y, 0.0, sizeof(double) * m);
			if (ellpack->fitsInMemory()) hipMemset(ellpack_y, 0.0, sizeof(double) * m);
		}
	}

	collectResults(csr, ellpack, csr_y, ellpack_y, n);
	cudaCheckError(__LINE__);

	deallocateCSR(csr_irp, csr_ja, csr_as, csr_x, csr_y);
	deallocateEllpack(ellpack_ja, ellpack_as, ellpack_x, ellpack_y, maxnz);
	cudaCheckError(__LINE__);
	
	io->exportResults(CUDA, path, csr, ellpack);
}
