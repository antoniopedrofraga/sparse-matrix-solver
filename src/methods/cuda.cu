#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

#include "../matrix/matrix.h"
#include "../matrix/csr.h"
#include "../matrix/ellpack.h"
#include "../io/iomanager.h"
#include "../utils/utils.h"


int thread_block = 512;


void cudaCheckError() {
	hipError_t e = hipGetLastError();
	if(e != hipSuccess) {
		printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));
		exit(0);
	}
}

__global__ void solveCSR(int * m, int * irp, int * ja, double * as, double * x, double * y) {
	extern __shared__ double sdata[];
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	/*int tid = threadIdx.x;

	int warp = i / 32; 
	int lane = i & (32 - 1); 

	int row = warp;
	sdata[tid] = 0;

	if (row < *m) {
		for (int j = irp[row] + lane ; j < irp[row + 1]; j += 32)
			sdata[tid] += as[j] * x[ja[j]];

		if (lane < 16) sdata[tid] += sdata[tid + 16];
		if (lane < 8) sdata[tid] += sdata[tid + 8];
		if (lane < 4) sdata[tid] += sdata[tid + 4];
		if (lane < 2) sdata[tid] += sdata[tid + 2];
		if (lane < 1) sdata[tid] += sdata[tid + 1];

		if (lane == 0)
			y[row] += sdata[tid];
	}*/
	if (i < *m) {
		double temp = 0.0;
		for (int j = irp[i]; j < irp[i + 1]; ++j) {
			temp += as[j] * x[ja[j]];
		}
		y[i] = temp;
	}
	__syncthreads();
}

__global__ void solveEllpack(int * m, int * ja, double * as, double * x, double * y, int * maxnz) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int rows = *m, maximum_nz = *maxnz;
	if (i < rows) {
		double temp = 0.0;
		for (int j = 0; j < *maxnz; ++j) {
			temp += as[i * maximum_nz + j] * x[ja[i * maximum_nz + j]];
		}
		y[i] = temp;
	}
	__syncthreads();
}

void allocateCSR(CSR * &csr, int * &irp, int * &ja, double * &as, double * &x, double *&y, int &m, int &n) {
	int nz = csr->getnz();

	hipMalloc((void**)&irp, sizeof(int) * (nz + 1));
	hipMalloc((void**)&ja, sizeof(int) * nz);
	hipMalloc((void**)&as, sizeof(double) * nz);
	hipMalloc((void**)&x, sizeof(double) * n);
	hipMalloc((void**)&y, sizeof(double) * n);
	hipMemcpy(irp, csr->getirp(), sizeof(int) * (nz + 1), hipMemcpyHostToDevice);
	hipMemcpy(ja, csr->getja(), sizeof(int) * nz, hipMemcpyHostToDevice);
	hipMemcpy(as, csr->getas(), sizeof(double) * nz, hipMemcpyHostToDevice);
	hipMemcpy(x, csr->getX(), sizeof(double) * n, hipMemcpyHostToDevice);
	hipMemcpy(y, csr->y, sizeof(double) * n, hipMemcpyHostToDevice);
}

void allocateEllpack(Ellpack * &ellpack, int * &ja, double * &as, double * &x, double * &y, int * &maxnz, int &m, int &n) {
	int host_maxnz = ellpack->getmaxnz();
	int * host_ja = ellpack->get1Dja();
	double * host_as = ellpack->get1Das();

	hipMalloc((void**)&ja, sizeof(int) * m * host_maxnz);
	hipMalloc((void**)&as,  sizeof(double) * m * host_maxnz);
	hipMalloc((void**)&x, sizeof(double) * m);
	hipMalloc((void**)&y, sizeof(double) * m);
	hipMalloc((void**)&maxnz, sizeof(int));
	hipMemcpy(ja, host_ja, sizeof(int) * m * host_maxnz, hipMemcpyHostToDevice);
	hipMemcpy(as, host_as, sizeof(double) * m * host_maxnz, hipMemcpyHostToDevice);
	hipMemcpy(x, ellpack->getX(), sizeof(double) * m, hipMemcpyHostToDevice);
	hipMemcpy(y, ellpack->y, sizeof(double) * m, hipMemcpyHostToDevice);
	hipMemcpy(maxnz, &host_maxnz, sizeof(int), hipMemcpyHostToDevice);
}

void collectResults(CSR * &csr, Ellpack * &ellpack, double * &csr_y, double * &ellpack_y, int &n) {
	hipMemcpy(csr->y, csr_y, sizeof(double) * n, hipMemcpyDeviceToHost);
	hipMemcpy(ellpack->y, ellpack_y, sizeof(double) * n, hipMemcpyDeviceToHost);
}


void solveCuda(IOmanager * io, std::string path, CSR * &csr, Ellpack * &ellpack) {
	
	int m = csr->getRows();
	int n = csr->getCols();
	const int shmem_size = thread_block * sizeof(double);

	int n_blocks = m / thread_block;

	if (m % thread_block > 0.0) {
		n_blocks++;
	}

	int * csr_irp, * csr_ja, * ellpack_ja, * maxnz, * rows;
	double * csr_as, * csr_x, * csr_y, * ellpack_as, * ellpack_x, * ellpack_y;
	
	allocateCSR(csr, csr_irp, csr_ja, csr_as, csr_x, csr_y, m, n);
	allocateEllpack(ellpack, ellpack_ja, ellpack_as, ellpack_x, ellpack_y, maxnz, m, n);

	hipMalloc((void**)&rows, sizeof(int));
	hipMemcpy(rows, &m, sizeof(int), hipMemcpyHostToDevice);

	
	for (int k = 0; k < NR_RUNS; ++k) {
		csr->trackTime();
		solveCSR<<<n_blocks, thread_block, shmem_size>>>(rows, csr_irp, csr_ja, csr_as, csr_x, csr_y);
		csr->trackTime();
		hipMemset(csr_y, 0, sizeof(double) * m);
		cudaCheckError();
		
		ellpack->trackTime();
		solveEllpack<<<n_blocks, thread_block>>>(rows, ellpack_ja, ellpack_as, ellpack_x, ellpack_y, maxnz);
		ellpack->trackTime();
		hipMemset(ellpack_y, 0, sizeof(double) * m);
		cudaCheckError();
	}

	collectResults(csr, ellpack, csr_y, ellpack_y, n);
	cudaCheckError();
	
	io->exportResults(CUDA, path, csr, ellpack);
}
