#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

#include "matrix/csr.h"
#include "matrix/ellpack.h"
#include "io/iomanager.h"
#include "matrix/matrix.h"
 
__global__ void solveCSR(CSR * csr) {
	int i = threadIdx.x;
	double t = 0.0;
	for (size_t k = 0; k < csr->irp_size; k++) {
		int j = csr->irp[k];
		t += csr->as[j] * csr->x[csr->ja[j]];
	}
	csr->y[i] = t;
}

__global__ void solveEllpack(Ellpack * ellpack) {
	int i = threadIdx.x;
	double t = 0.0;
	for (int j = 0; j < ellpack->maxnz; j++) {
		t += ellpack->as[i][j] * ellpack->x[ellpack->ja[i][j]];
	}
	ellpack->y[i] = t;
}

void solveCuda(CSR * csr, Ellpack * ellpack) {
	
	const int m = csr->getCols();
	const int csize = sizeof(CSR);
	const int esize = sizeof(Ellpack);
	
	CSR * csr_c;
	Ellpack * ellpack_c;

	hipMalloc((void**)&csr_c, csize);
	hipMalloc((void**)&ellpack_c, esize);
	hipMemcpy(csr_c, csr, csize, hipMemcpyHostToDevice); 
	hipMemcpy(ellpack_c, ellpack, esize, hipMemcpyHostToDevice); 
	
	solveCSR<<<1, m>>>(csr_c);
	solveEllpack<<<1, m>>>(ellpack_c);
	
	hipMemcpy(csr, csr_c, csize, hipMemcpyDeviceToHost); 
	hipMemcpy(ellpack, ellpack_c, esize, hipMemcpyDeviceToHost); 
	hipFree(csr_c);
	hipFree(ellpack_c);

	/*char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 
	char *ad;
	int *bd;
	const int csize = N * sizeof(char);
	const int isize = N * sizeof(int);
 
	printf("%s", a);
 
	hipMalloc((void**)&ad, csize); 
	hipMalloc((void**)&bd, isize); 
	hipMemcpy(ad, a, csize, hipMemcpyHostToDevice); 
	hipMemcpy(bd, b, isize, hipMemcpyHostToDevice); 
  
	dim3 dimGrid(25, 25);
	dim3 dimBlock(10, 10);
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy(a, ad, csize, hipMemcpyDeviceToHost); 
	hipFree(ad);
	hipFree(bd);

	std::cout << a << std::endl;*/
}
